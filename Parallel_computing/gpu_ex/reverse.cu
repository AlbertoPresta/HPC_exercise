
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define N 100
#define THREADS_PER_BLOCKS 10

//space for functions

__global__ void reverseArrayBlock(int *dev_a,int *dev_b){
        int bx = blockIdx.x;
        int tx = threadIdx.x;
        int old_id = blockDim.x*bx + tx;
        int new_id = (blockDim.x*gridDim.x) - 1 - old_id;

        dev_b[old_id] = dev_a[new_id];
}





int main(void){

        //pointer for host memory
        int *h_a;
        //pointer for device memory
        int *dev_a;
        int *dev_b;

        // compute number of blocks needed
        int numblocks = N/THREADS_PER_BLOCKS;

        //allocate host and device memory
        int mem_size = N*(sizeof(int));
        h_a = (int*)malloc(mem_size);
        hipMalloc((void**)&dev_a,mem_size);
        hipMalloc((void**)&dev_b,mem_size);
        //initialize array
        for (int i = 0;i<N;i++){h_a[i]=i;}

        // copy host array yo device array (in order to launch kernel)

        hipMemcpy(dev_a,h_a,mem_size,hipMemcpyHostToDevice);


        //launch kernel

        reverseArrayBlock<<<N/THREADS_PER_BLOCKS,THREADS_PER_BLOCKS>>>(dev_a,dev_b);


        //device to host copy
		hipMemcpy(h_a,dev_b,mem_size,hipMemcpyDeviceToHost);

		        for (int i =0; i<N;i++){printf("%d ",h_a[i]);}
		        // free davice memory
		        hipFree(dev_a);
		        hipFree(dev_b);

		        //free host_memory
		        free(h_a);
		        return 0;
		}
		