
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>


#define N 10
#define N_THREADS 2

// space for function

__global__ void mat_transpose(int *mat_in_dev, int *mat_out_dev){
        int index = threadIdx.x  +  blockIdx.x*blockDim.x;
        int x = index%N;
        int y = index/N;
        mat_out_dev[y*N+ x] = mat_in_dev[x*N+y];
        }

void fill_matrix(int *p){
        for(int i = 0; i<N;i++){
                for(int j = 0; j<N;j++){p[i*N + j] = rand()%100 + 1;}
                }
        }

void print_matrix(int *p){
        for(int i = 0;i<N;i++){
                for(int j=0;j<N;j++){printf("%d ",p[i*N + j]);}
                printf("\n");
}
}


int main(void){

        int *mat_in_h;
        int *mat_out_h;
        int *mat_in_dev;
        int *mat_out_dev;
		
		
		// request of host memory

		 mat_in_h = (int *)malloc(N*N*sizeof(int));
		 mat_out_h = (int *)malloc(N*N*sizeof(int));

		 // request of device memory

		 hipMalloc((void**)&mat_in_dev,N*N*sizeof(int));
		 hipMalloc((void**)&mat_out_dev,N*N*sizeof(int));

		 // inizialization of the matrix
		 fill_matrix(mat_in_h);
		 int size = N*N*sizeof(int);
		 // copy from host memomy to device memory

		 hipMemcpy(mat_in_dev,mat_in_h,size,hipMemcpyHostToDevice);
		
	      // kernel
		  int N_BLOCKS = N*N/N_THREADS;
          mat_transpose<<<N_BLOCKS,N_THREADS>>>(mat_in_dev,mat_out_dev);

		   //copy from device memory to host memory

		   hipMemcpy(mat_out_h,mat_out_dev,size,hipMemcpyDeviceToHost);
				       
				       
		    // free the memory

		    free(mat_in_h);
			free(mat_out_h);
	        hipFree(mat_in_dev);
			hipFree(mat_out_dev);
			return 0;










			}

			
