
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>


#define N 8192
#define N_THREADS 64

// space for function

__global__ void mat_transpose(int *mat_in_dev, int *mat_out_dev){
        int index = threadIdx.x  +  blockIdx.x*blockDim.x;
        int x = index%N;
        int y = index/N;
        mat_out_dev[y*N+ x] = mat_in_dev[x*N+y];
        }

void fill_matrix(int *p){
        for(int i = 0; i<N;i++){
                for(int j = 0; j<N;j++){p[i*N + j] = rand()%100 + 1;}
                }
        }

void print_matrix(int *p){
        for(int i = 0;i<N;i++){
                for(int j=0;j<N;j++){printf("%d ",p[i*N + j]);}
                printf("\n");
}
}


int main(void){

        int *mat_in_h;
        int *mat_out_h;
        int *mat_in_dev;
        int *mat_out_dev;
		
		
		// request of host memory

		 mat_in_h = (int *)malloc(N*N*sizeof(int));
		 mat_out_h = (int *)malloc(N*N*sizeof(int));

		 // request of device memory

		 hipMalloc((void**)&mat_in_dev,N*N*sizeof(int));
		 hipMalloc((void**)&mat_out_dev,N*N*sizeof(int));

		 // inizialization of the matrix
		 fill_matrix(mat_in_h);
		 int size = N*N*sizeof(int);
		 // copy from host memomy to device memory

	         hipMemcpy(mat_in_dev,mat_in_h,size,hipMemcpyHostToDevice);


		hipEvent_t start, stop;
	  	hipEventCreate(&start);
		hipEventCreate(&stop);

		
	      // kernel
		int N_BLOCKS = N*N/N_THREADS;
		hipEventRecord(start);                
                mat_transpose<<<N_BLOCKS,N_THREADS>>>(mat_in_dev,mat_out_dev);
		hipEventRecord(stop);
		//copy from device memory to host memory

		hipMemcpy(mat_out_h,mat_out_dev,size,hipMemcpyDeviceToHost);
				       
		hipEventSynchronize(stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("dimensions of block: %d x %d\n",8,8);
        	printf(" Time in milliseconds: %f\n",milliseconds);
        	printf("Bandwidth: %f GB/s\n",2*size/milliseconds/1e6);		
		
		
						       
		    // free the memory

		free(mat_in_h);
		free(mat_out_h);
	        hipFree(mat_in_dev);
		hipFree(mat_out_dev);
		return 0;

	}

			
