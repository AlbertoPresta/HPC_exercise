
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>

#define N 8192
#define LINEAR_SIDE 32

void print_matrix(int *p){
        for(int i = 0;i<N;i++){
                for(int j=0;j<N;j++){printf("%d ",p[i*N + j]);}
                printf("\n");
}
}


void fill_matrix(int *p){
        for(int i = 0; i<N;i++){
           for(int j = 0; j<N;j++){p[i*N + j] = rand()%100 + 1;}
           }
 }

bool verify(int *a,int*b){
	
	for(int i =0;i<N*N;i++){if(a[i]!=b[(i%N)*N + i/N]) return 0;}
	return 1;	
}


__global__ void transpose(int *mat_in_dev, int *mat_out_dev){

	//local memory location where to save  matrix portion
	//related to the blocks
	// shared by all threads within a block
	__shared__  int temp_matrix[LINEAR_SIDE][LINEAR_SIDE];
	
//	creation of the global indexes in order to journey to the matrix 	
	int global_x = blockIdx.x*blockDim.x + threadIdx.x;
	int global_y = blockIdx.y*blockDim.y + threadIdx.y;
	
//	now we created global indexes referred to transpose matrix

	//copy in the sub matrix
	temp_matrix[threadIdx.x][threadIdx.y] = mat_in_dev[global_x*N + global_y];
	
	__syncthreads();
	
	//copy submatrix in out
	
	mat_out_dev[global_y*N + global_x]=temp_matrix[threadIdx.x][threadIdx.y];
	
}



                                                
int main(void){
	
	int *mat_in_h;
	int *mat_out_h;
	int *mat_in_dev;
	int *mat_out_dev;

	//allocation of memory
	mat_in_h = (int*)malloc(N*N*sizeof(int));
	mat_out_h = (int*)malloc(N*N*sizeof(int));
	hipMalloc((void**)&mat_in_dev,N*N*sizeof(int));
	hipMalloc((void**)&mat_out_dev,N*N*sizeof(int));
	fill_matrix(mat_in_h);	

	//copy matrix from host to device
	int size = N*N*sizeof(int);
	hipMemcpy(mat_in_dev,mat_in_h,size,hipMemcpyHostToDevice);
	
	// definition of variables dim3
	dim3 grid,block;
	
	block.x = LINEAR_SIDE;
	block.y = LINEAR_SIDE;
	grid.x = N/block.x;
	grid.y = N/block.y;

//	timing
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);



	hipEventRecord(start);
	transpose<<<grid,block>>>(mat_in_dev,mat_out_dev);
	hipEventRecord(stop);	

	// copy from device to host
	
	hipMemcpy(mat_out_h,mat_out_dev,size,hipMemcpyDeviceToHost); 
	printf("%d\n",verify(mat_in_h,mat_out_h));

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("dimensions of block: %d x %d\n",LINEAR_SIDE,LINEAR_SIDE);
        printf(" Time in milliseconds: %f\n",milliseconds);
        printf("Bandwidth: %f GB/s\n",2*size/milliseconds/1e6);


	//free the memory
	free(mat_in_h);
	free(mat_out_h);
	hipFree(mat_in_dev);
	hipFree(mat_out_dev);
	
	
	return 0;
}
